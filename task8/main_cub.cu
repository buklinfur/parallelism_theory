#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <cmath>
#include <vector>

#include <hipcub/hipcub.hpp>
#include <boost/program_options.hpp>
#include <hip/hip_runtime.h>

#define OFFSET(i, j, N) ((i) * (N) + (j))
constexpr int BLOCK_SIZE = 1024;

__global__ void calculate_step(int N,
                               const double *__restrict__ in,
                               double *__restrict__ out,
                               double *block_errors) {
    using BlockReduce = hipcub::BlockReduce<double, BLOCK_SIZE>;
    __shared__ typename BlockReduce::TempStorage tmp;  

    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int i = gid / N, j = gid % N;
    double local_err = 0.0;

    if (i > 0 && i < N - 1 && j > 0 && j < N - 1)
    {
        double v = (in[OFFSET(i - 1, j, N)] +
                    in[OFFSET(i + 1, j, N)] +
                    in[OFFSET(i, j - 1, N)] +
                    in[OFFSET(i, j + 1, N)] +
                    in[OFFSET(i, j, N)]) *
                   0.2;
        out[OFFSET(i, j, N)] = v;
        local_err = fabs(v - in[OFFSET(i, j, N)]); 
    }

    double block_max = BlockReduce(tmp).Reduce(local_err, hipcub::Max());
    if (threadIdx.x == 0)
    {
        block_errors[blockIdx.x] = block_max; 
    }
}

__global__ void reduce_global(int num_blocks, double *block_errors) {
    using BlockReduce = hipcub::BlockReduce<double, BLOCK_SIZE>;
    __shared__ typename BlockReduce::TempStorage tmp;

    int t = threadIdx.x;
    double v = (t < num_blocks ? block_errors[t] : 0.0);
    double m = BlockReduce(tmp).Reduce(v, hipcub::Max());
    if (t == 0)
        block_errors[0] = m;
}

void init_boundary(std::vector<double> &A, int N)
{
    double TL = 10, TR = 20, BR = 30, BL = 20;

    for (int j = 0; j < N; ++j) {
        double t = double(j) / (N - 1);
        A[OFFSET(0, j, N)] = (1 - t) * TL + t * TR;
    }

    for (int i = 0; i < N; ++i) {
        double t = double(i) / (N - 1);
        A[OFFSET(i, N - 1, N)] = (1 - t) * TR + t * BR;
    }

    for (int j = 0; j < N; ++j) {
        double t = double(j) / (N - 1);
        A[OFFSET(N - 1, j, N)] = (1 - t) * BR + t * BL;
    }

    for (int i = 0; i < N; ++i) {
        double t = double(i) / (N - 1);
        A[OFFSET(i, 0, N)] = (1 - t) * BL + t * TL;
    }
}

int main(int argc, char *argv[])
{
    namespace po = boost::program_options;
    int N;
    double eps;
    int max_iters;
    bool draw;

    po::options_description desc("Options");
    desc.add_options()
        ("help,h", "help")
        ("size,s", po::value<int>(&N)->default_value(512), "grid size")
        ("max_error,me", po::value<double>(&eps)->default_value(1e-6), "tolerance")
        ("max_iterations,mi", po::value<int>(&max_iters)->default_value(1'000'000), "max iters")
        ("draw_output,do", po::bool_switch(&draw), "print final mat");
    po::variables_map vm;
    po::store(po::parse_command_line(argc, argv, desc), vm);
    po::notify(vm);

    if (vm.count("help")) {
        std::cout << desc << "\n";
        return 0;
    }

    std::vector<double> h_A(N * N, 0.0), h_B;
    init_boundary(h_A, N);
    h_B = h_A;

    double *d_in, *d_out, *d_err;
    hipMalloc(&d_in, N * N * sizeof(double));
    hipMalloc(&d_out, N * N * sizeof(double));
    int num_threads = N * N;
    int blocks = (num_threads + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hipMemcpy(d_in, h_A.data(), N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_out, h_B.data(), N * N * sizeof(double), hipMemcpyHostToDevice);

    double error = 1.0;
    int iter = 0;
    auto t0 = std::chrono::steady_clock::now();

    while (error > eps && iter < max_iters) {
        hipGraph_t graph;
        hipGraphExec_t graphExec;

        hipStream_t s;
        hipStreamCreate(&s);
        hipStreamBeginCapture(s, hipStreamCaptureModeGlobal);
        
        calculate_step<<<blocks, BLOCK_SIZE, 0, s>>>(N, d_in, d_out, d_err);

        hipStreamEndCapture(s, &graph);
        hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0); 

        hipGraphLaunch(graphExec, s);
        hipStreamSynchronize(s);

        hipGraphExecDestroy(graphExec);
        hipGraphDestroy(graph);
        hipStreamDestroy(s);

        reduce_global<<<1, BLOCK_SIZE>>>(blocks, d_err);
        hipMemcpy(&error, d_err, sizeof(double), hipMemcpyDeviceToHost);

        std::swap(d_in, d_out);
        ++iter;
    }
    hipDeviceSynchronize();
    auto t1 = std::chrono::steady_clock::now();

    std::cout << "Iterations:   " << iter << "\n"
              << "Final error:  " << error << "\n"
              << "Elapsed time: " << std::chrono::duration<double>(t1 - t0).count()
              << " s\n";

    if (draw) {
        hipMemcpy(h_B.data(), d_out, N * N * sizeof(double), hipMemcpyDeviceToHost);
        for (int i = 0; i < N; ++i) {
            for (int j = 0; j < N; ++j)
                std::cout << h_B[OFFSET(i, j, N)] << " ";
            std::cout << "\n";
        }
    }

    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_err);
    return 0;
}